#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <sys/time.h>

#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

void cudaCheck_f(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

bool floateq(float a, float b, float eps){
  return abs(a - b) < eps;
}

#define cudaCheck(err) (cudaCheck_f(err, __FILE__, __LINE__))

__device__ void compute(float* global_out, float shared_in) {
    // Computes using all values of current batch from shared memory.
    // Stores this thread's result back to global memory.
    // printf("COMPUTE: %d => %lf (s)\n", threadIdx.x, shared_in[threadIdx.x]);
    *(global_out + threadIdx.x) = 6.9f * shared_in + 4.20f;
}

__global__ void without_memcpy_async(float* global_out, float const* global_in, size_t size, size_t batch_sz) {
  auto grid = cooperative_groups::this_grid();
  auto block = cooperative_groups::this_thread_block();
  assert(size == batch_sz * grid.size()); // Exposition: input size fits batch_sz * grid_size

  __shared__ float shared[1024]; // block.size() * sizeof(float) bytes

  #pragma unroll 1
  for (size_t batch = 0; batch < batch_sz; ++batch) {
    // Compute the index of the current batch for this block in global memory:
    size_t block_batch_idx = batch * blockDim.x;
    size_t global_idx = block_batch_idx + threadIdx.x;
    shared[threadIdx.x] = global_in[global_idx];

    block.sync(); // Wait for all copies to complete
    // printf("%d => %lf (g), %lf (s)\n", threadIdx.x, global_in[global_idx], shared[local_idx]);
    compute(global_out + block_batch_idx, shared[threadIdx.x]); // Compute and write result to global memory
    block.sync(); // Wait for compute using shared memory to finish
  }
}


// __global__ void with_memcpy_async(float* global_out, float const* global_in, size_t size, size_t batch_sz) {
//   auto grid = cooperative_groups::this_grid();
//   auto block = cooperative_groups::this_thread_block();
//   assert(size == batch_sz * grid.size()); // Exposition: input size fits batch_sz * grid_size

//   __align__(16) __shared__ float shared[1024]; // block.size() * sizeof(float) bytes

//   // Whole thread-group cooperatively copies whole batch to shared memory:
//   #pragma unroll
//   for (size_t batch = 0; batch < batch_sz; ++batch) {
//     size_t block_batch_idx = grid.size() * batch;
//     cooperative_groups::memcpy_async(block, shared, global_in + block_batch_idx, cuda::aligned_size_t<16>(sizeof(float) * block.size()));
//     cooperative_groups::wait(block); // Joins all threads, waits for all copies to complete
//     compute(global_out + block_batch_idx, shared[threadIdx.x]);
//     block.sync();
//   }
// }

template<int block_dim, int num_stages>
__global__ void pipelined(float* dest, float const* src, size_t size) {
    // Read blockDim.x integers per pipeline stage
   __shared__  float smem[num_stages][block_dim];
 
    // Grid stride loop:
    int offset = blockIdx.x * blockDim.x;
    size_t stride = gridDim.x * blockDim.x;
 
    // No pipeline::shared_state needed
    // cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
 
    // Load all pipeline stages.
    for (int stage = 0; stage < num_stages; ++stage) {
        // pipe.producer_acquire();
        size_t idx = offset + stage * stride + threadIdx.x;
        if (idx < size) {
          asm volatile(
            "cp.async.ca.shared.global [%0], [%1], %2, %3;\n"
            :
            : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&smem[stage][threadIdx.x]))),
              "l"(&src[idx]),
              "n"(sizeof(float)), "n"(sizeof(float))
            : "memory"
          );
          // cuda::memcpy_async(&smem[stage][threadIdx.x], &src[idx], sizeof(float), pipe);
        }
        asm volatile("cp.async.commit_group;");
        // pipe.producer_commit();
    }

    // At this point, there are `num_stages` commited into the pipeline. This is a loop.
    // invariant that is upheld throughout the loop.
    int stage = 0;
    for (size_t block_idx = offset; block_idx < size; block_idx += stride) {
        // Wait for the first stage to have completed loading, or equivalently: wait until
        // at most `num_stages - 1` stages are still loading.
        // cuda::pipeline_consumer_wait_prior<num_stages - 1>(pipe);
        asm volatile("cp.async.wait_group 1;");

        // __syncthreads is necessary if other threads want to read this thread's loaded data.
        // __syncthreads();
        compute(dest + block_idx, smem[stage][threadIdx.x]);
        __syncthreads();

        // Release the consumed stage.
        // pipe.consumer_release();

        // Pre-load data for `num_stages` into the future.
        // pipe.producer_acquire();
        // To ensure that the number of commited stages into the pipeline remains constant,
        // producer_acquire and producer_commit are called even if the load is out-of-bounds.
        size_t idx = block_idx + num_stages * stride + threadIdx.x;
        if (idx < size) {
              asm volatile(
                "cp.async.ca.shared.global [%0], [%1], %2, %3;\n"
                :
                : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&smem[stage][threadIdx.x]))),//  &smem[stage][threadIdx.x]), 
                  "l"(&src[idx]),
                  "n"(sizeof(float)), "n"(sizeof(float))
                : "memory"
            );
            // cuda::memcpy_async(&smem[stage][threadIdx.x], &src[idx], sizeof(float), pipe);
        }
        asm volatile("cp.async.commit_group;");
        // pipe.producer_commit();

        stage = (stage + 1) % num_stages;
    }
}


void randomize_matrix(float *mat, int N) {
  // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
  // precision is too low and the same random number is generated.
  struct timeval time {};
  gettimeofday(&time, nullptr);
  for (int i = 0; i < N; i++) {
    float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
    tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
    mat[i] = tmp;
  }
}


int main(int argc, char **argv) {
  float *Xs, *Ys;
  float *Xs_d, *Ys_d;

  uint SIZE = 4096;

  Xs = (float *)malloc(SIZE * SIZE * sizeof(float));
  Ys = (float *)malloc(SIZE * SIZE * sizeof(float));

  cudaCheck(hipMalloc((void **)&Xs_d, SIZE * SIZE * sizeof(float)));
  cudaCheck(hipMalloc((void **)&Ys_d, SIZE * SIZE * sizeof(float)));

  randomize_matrix(Xs, SIZE*SIZE);

  cudaCheck(hipMemcpy(Xs_d, Xs, SIZE*SIZE*sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipGetLastError()); // Check for async errors during kernel run

  float elapsed_time;
  float num_times = 50;
  hipEvent_t beg, end;
  cudaCheck(hipEventCreate(&beg));
  cudaCheck(hipEventCreate(&end));
  cudaCheck(hipEventRecord(beg));
  for (int j = 0; j < num_times; j++) {
    // with_memcpy_async<<<1, 1024>>>(Ys_d, Xs_d, SIZE * SIZE, (SIZE*SIZE)/1024);
    pipelined<1024, 2><<<1, 1024>>>(Ys_d, Xs_d, SIZE*SIZE);
  }
  cudaCheck(hipEventRecord(end));
  cudaCheck(hipEventSynchronize(beg));
  cudaCheck(hipEventSynchronize(end));
  cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));
  elapsed_time /= 1000.; // Convert to seconds

  printf(
      "Average elapsed time: (%7.6f) s.\n\n", elapsed_time / num_times);
  fflush(stdout);

  cudaCheck(hipMemcpy(Ys, Ys_d, SIZE * SIZE * sizeof(float), hipMemcpyDeviceToHost));

  for(int i=0;i<SIZE*SIZE;i++){
    if(!floateq(Ys[i], 6.9f * Xs[i] + 4.20f, 0.001f)){
      printf("Validation Failed (%d)! %lf => %lf\n", i, Xs[i], Ys[i]);
      break;
    }
  }
  printf("Validation Pass!\n");

  cudaCheck(hipDeviceSynchronize());

  hipFree(Xs_d);
  hipFree(Ys_d);
  free(Xs);
  free(Ys);
}
