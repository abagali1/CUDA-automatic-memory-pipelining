#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <vector>

#include <time.h>
#include <unistd.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define cudaCheck(err) (cudaCheckf(err, __FILE__, __LINE__))

extern "C" {
  extern unsigned char kernel_fatbin[];
  extern unsigned int kernel_fatbin_len;
}

const std::string errLogFile = "matrixValidationFailure.txt";
float get_sec() {
  struct timeval time;
  gettimeofday(&time, NULL);
  return (1e6 * time.tv_sec + time.tv_usec);
}

float cpu_elapsed_time(float &beg, float &end) { return 1.0e-6 * (end - beg); }

void cudaCheckf(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

void randomize_matrix(float *mat, int N) {
  // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
  // precision is too low and the same random number is generated.
  struct timeval time {};
  gettimeofday(&time, nullptr);
  for (int i = 0; i < N; i++) {
    float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
    tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
    mat[i] = tmp;
  }
}

void range_init_matrix(float *mat, int N) {
  for (int i = 0; i < N; i++) {
    mat[i] = i;
  }
}

void zero_init_matrix(float *mat, int N) {
  for (int i = 0; i < N; i++) {
    mat[i] = 0.0;
  }
}

void copy_matrix(const float *src, float *dest, int N) {
  int i;
  for (i = 0; src + i && dest + i && i < N; i++)
    *(dest + i) = *(src + i);
  if (i != N)
    printf("copy failed at %d while there are %d elements in total.\n", i, N);
}

void print_matrix(const float *A, int M, int N, std::ofstream &fs) {
  int i;
  fs << std::setprecision(2)
     << std::fixed; // Set floating-point precision and fixed notation
  fs << "[";
  for (i = 0; i < M * N; i++) {
    if ((i + 1) % N == 0)
      fs << std::setw(5) << A[i]; // Set field width and write the value
    else
      fs << std::setw(5) << A[i] << ", ";
    if ((i + 1) % N == 0) {
      if (i + 1 < M * N)
        fs << ";\n";
    }
  }
  fs << "]\n";
}

bool verify_matrix(float *matRef, float *matOut, int N) {
  double diff = 0.0;
  int i;
  for (i = 0; i < N; i++) {
    diff = std::fabs(matRef[i] - matOut[i]);
    if (diff > 0.015) {
      printf("Divergence! Should %lf, Is %lff (Diff %lff) at %d\n",
             matRef[i], matOut[i], diff, i);
      return false;
    }
  }
  return true;
}

int div_ceil(int numerator, int denominator) {
  std::div_t res = std::div(numerator, denominator);
  return res.rem ? (res.quot + 1) : res.quot;
}

extern "C" __global__ void sgemm_naive(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C);
extern "C" __global__ void sgemm_global_mem_coalesce(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C);
extern "C" __global__ void sgemm_shared_mem_block(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C);
extern "C" __global__ void sgemm_shared_mem_block_async(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C);
extern "C" __global__ void sgemm_shared_mem_block_async_overlap(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C);


void run_sgemm_naive(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}
  
void run_sgemm_coalesce(int M, int N, int K, float alpha, float *A, float *B,
       float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  sgemm_global_mem_coalesce<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}
  
void run_sgemm_shared_mem_block(int M, int N, int K, float alpha, float *A,
               float *B, float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  // L1 cache becomes useless, since we access GMEM only via SMEM, so we carve
  // out all of L1 to SMEM. This doesn't currently make a difference, since
  // occupancy is limited by reg and thread count, but it's good to do anyway.
  hipFuncSetAttribute(reinterpret_cast<const void*>(sgemm_shared_mem_block),
      hipFuncAttributePreferredSharedMemoryCarveout,
      cudaSharedmemCarveoutMaxShared);
  sgemm_shared_mem_block<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}
  
void run_sgemm_shared_mem_block_async(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C){
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  
  sgemm_shared_mem_block_async<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}
  
void run_sgemm_shared_mem_block_async_overlap(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C){
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  // L1 cache becomes useless, since we access GMEM only via SMEM, so we carve
  // out all of L1 to SMEM. This doesn't currently make a difference, since
  // occupancy is limited by reg and thread count, but it's good to do anyway.
  hipFuncSetAttribute(reinterpret_cast<const void*>(sgemm_shared_mem_block),
      hipFuncAttributePreferredSharedMemoryCarveout,
      cudaSharedmemCarveoutMaxShared);
  sgemm_shared_mem_block_async_overlap<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}


void run_kernel(int kernel_num, int M, int N, int K, float alpha, float *A,
                float *B, float beta, float *C) {
  switch (kernel_num) {
  case 0:
    run_sgemm_naive(M, N, K, alpha, A, B, beta, C);
    break;
  case 1:
    run_sgemm_coalesce(M, N, K, alpha, A, B, beta, C);
    break;
  case 2:
    run_sgemm_shared_mem_block(M, N, K, alpha, A, B, beta, C);
    break;
  case 3:
    run_sgemm_shared_mem_block_async(M, N, K, alpha, A, B, beta, C);
    break;
  case 4:
    run_sgemm_shared_mem_block_async_overlap(M, N, K, alpha, A, B, beta, C);
    break;
  default:
    std::cout << "BAD: " << kernel_num << std::endl;
    throw std::invalid_argument("Unknown kernel number");
  }
}

void dump_mat(const std::string fname, const float *arr, const size_t size){
  FILE *f = fopen(fname.c_str(), "w");
  for(int i=0;i<size;i++){
    for(int j=0;j<size;j++){
      fprintf(f, "%lf ", arr[i * size + j]);
    }
    fprintf(f, "\n");
  }
  fclose(f);
}

int main(int argc, char **argv) {
  if (argc != 2) {
    std::cerr << "Please select a kernel (range 0 - 12, 0 for NVIDIA cuBLAS)"
              << std::endl;
    exit(EXIT_FAILURE);
  }

  // get kernel number
  int kernel_num = std::stoi(argv[1]);
  if (kernel_num < 0 || kernel_num > 12) {
    std::cerr << "Please enter a valid kernel number (0-12)" << std::endl;
    exit(EXIT_FAILURE);
  }

  // get environment variable for device
  int deviceIdx = 0;
  if (getenv("DEVICE") != NULL) {
    deviceIdx = atoi(getenv("DEVICE"));
  }
  cudaCheck(hipSetDevice(deviceIdx));

  printf("Running kernel %d on device %d.\n", kernel_num, deviceIdx);

  // print some device info
  // CudaDeviceInfo();

  // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
  // publishing event tasks in the target stream
  float elapsed_time;
  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);

  // cuBLAS FLOPs ceiling is reached at 8192
  std::vector<int> SIZE = {64, 128, 256, 512, 1024, 2048, 4096, 8192};// 16384};

  long m, n, k, max_size;
  max_size = SIZE[SIZE.size() - 1];
  std::cout << "Max size: " << max_size << std::endl;

  float alpha = 1.0f, beta = 1.0f; // GEMM input parameters, C=α*AB+β*C

  float *A = nullptr, *B = nullptr, *C = nullptr,
        *C_ref = nullptr; // host matrices
  float *dA = nullptr, *dB = nullptr, *dC = nullptr,
        *dC_ref = nullptr; // device matrices

  A = (float *)malloc(sizeof(float) * max_size * max_size);
  B = (float *)malloc(sizeof(float) * max_size * max_size);
  C = (float *)malloc(sizeof(float) * max_size * max_size);
  C_ref = (float*)malloc(sizeof(float) * max_size * max_size);

  randomize_matrix(A, max_size * max_size);
  randomize_matrix(B, max_size * max_size);

  for(int i=0;i<max_size*max_size;i++){
    C[i] = 0.0f;
    C_ref[i] = 0.0f;
  }

  cudaCheck(hipMalloc((void **)&dA, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dB, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dC, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dC_ref, sizeof(float) * max_size * max_size));

  cudaCheck(hipMemcpy(dA, A, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dB, B, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dC, C, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dC_ref, C, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));

  int repeat_times = 50;
  for (int size : SIZE) {
    m = n = k = size;

    std::cout << "dimensions(m=n=k) " << m << ", alpha: " << alpha
              << ", beta: " << beta << std::endl;
    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (kernel_num != 0) {
      run_kernel(0, m, n, k, alpha, dA, dB, beta, dC_ref); // Executes the kernel, modifies the result matrix
      run_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC); // Executes the kernel, modifies the result matrix

      cudaCheck(hipDeviceSynchronize());
      cudaCheck(hipGetLastError()); // Check for async errors during kernel run
      hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
      hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);

      if (m < 128) {
          std::cout << " Logging dummy check output into " << errLogFile << "\n";
          std::ofstream fs;
          fs.open(errLogFile);
          fs << "A:\n";
          print_matrix(A, m, n, fs);
          fs << "B:\n";
          print_matrix(B, m, n, fs);
          fs << "C:\n";
          print_matrix(C, m, n, fs);
          fs << "Should:\n";
          print_matrix(C_ref, m, n, fs);
      }
      if (!verify_matrix(C_ref, C, m * n)) {
        std::cout
            << "Failed to pass the correctness verification"
            << std::endl;
        exit(EXIT_FAILURE);
      }
    }

    hipEventRecord(beg);
    for (int j = 0; j < repeat_times; j++) {
      // We don't reset dC between runs to save time
      run_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC);
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.; // Convert to seconds

    long flops = 2 * m * n * k;
    printf(
        "Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS. size: "
        "(%ld).\n",
        elapsed_time / repeat_times,
        (repeat_times * flops * 1e-9) / elapsed_time, m);
    fflush(stdout);
    // make dC and dC_ref equal again (we modified dC while calling our kernel
    // for benchmarking)
    cudaCheck(hipMemcpy(dC, dC_ref, sizeof(float) * m * n,
                         hipMemcpyDeviceToDevice));
  }

  // Free up CPU and GPU space
  free(A);
  free(B);
  free(C);
  free(C_ref);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dC_ref);

  return 0;
};