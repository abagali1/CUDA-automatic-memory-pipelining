#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

#define BLOCKSIZE 32
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

namespace cg = cooperative_groups;

extern "C" __global__ void sgemm_naive(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C) {
    const uint y = blockIdx.x * blockDim.x + threadIdx.x;
    const uint x = blockIdx.y * blockDim.y + threadIdx.y;

    // if statement is necessary to make things work under tile quantization
    if (x < M && y < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
            tmp += A[x * K + i] * B[i * N + y];
        }
        // C = α*(A@B)+β*C
        C[x * N + y] = tmp; 
    }
}


extern "C" __global__ void sgemm_global_mem_coalesce(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C) {
    // const int cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    // const int cCol = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);
    const int cRow = blockIdx.y * BLOCKSIZE + threadIdx.y;
    const int cCol = blockIdx.x * BLOCKSIZE + threadIdx.x;

    // if statement is necessary to make things work under tile quantization
    if (cRow < M && cCol < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
            tmp += A[cRow * K + i] * B[i * N + cCol];
        }
        C[cRow * N + cCol] = tmp;
    }
}

extern "C" __global__ void sgemm_shared_mem_block(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C) {
    // the output block that we want to compute in this threadblock
    const uint cRow = blockIdx.x;
    const uint cCol = blockIdx.y;

    // allocate buffer for current block in fast shared mem
    // shared mem is shared between all threads in a block
    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];

    // the inner row & col that we're accessing in this thread
    const uint threadCol = threadIdx.x;// % BLOCKSIZE;
    const uint threadRow = threadIdx.y;// / BLOCKSIZE;

    // advance pointers to the starting positions
    // A += cRow * BLOCKSIZE * K;                    // row=cRow, col=0
    // B += cCol * BLOCKSIZE;                        // row=0, col=cCol
    C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol

    float tmp = 0.0;
    int num_batches = M / BLOCKSIZE;
    for (int bkIdx = 0; bkIdx < num_batches; bkIdx += 1) {
        // Have each thread load one of the elements in A & B
        // Make the threadCol (=threadIdx.x) the consecutive index
        // to allow global memory access coalescing

        As[threadRow * BLOCKSIZE + threadCol] = A[(cRow * BLOCKSIZE * K) + (BLOCKSIZE * bkIdx) + threadRow * K + threadCol];
        Bs[threadRow * BLOCKSIZE + threadCol] = B[(cCol * BLOCKSIZE) + (BLOCKSIZE * N * bkIdx) + threadRow * N + threadCol];

        // block threads in this block until cache is fully populated
        __syncthreads();
        // A += BLOCKSIZE;
        // B += BLOCKSIZE * N;

        // execute the dotproduct on the currently cached block
        for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
            tmp += As[threadRow * BLOCKSIZE + dotIdx] *
            Bs[dotIdx * BLOCKSIZE + threadCol];
        }
            // need to sync again at the end, to avoid faster threads
            // fetching the next block into the cache before slower threads are done
        __syncthreads();
    }
    C[threadRow * N + threadCol] = tmp;
}

extern "C" __global__ void sgemm_shared_mem_block_async(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C) {
    // the output block that we want to compute in this threadblock
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;

    auto group = cg::this_thread_block();
    auto tile = cg::tiled_partition<BLOCKSIZE>(group);

    // allocate buffer for current block in fast shared mem
    // shared mem is shared between all threads in a block
    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];


    // the inner row & col that we're accessing in this thread
    const uint threadCol = threadIdx.x;
    const uint threadRow = threadIdx.y;

    // advance pointers to the starting positions
    // A += cRow * BLOCKSIZE * K;                    // row=cRow, col=0
    // B += cCol * BLOCKSIZE;                        // row=0, col=cCol
    C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol

    float tmp = 0.0;
    // group.sync();
    size_t num_batches = K / BLOCKSIZE;
    for (int bkIdx = 0; bkIdx < num_batches ; bkIdx += 1) {
        // Have each thread load one of the elements in A & B
        // Make the threadCol (=threadIdx.x) the consecutive index
        // to allow global memory access coalescing
        // __syncthreads();
        // As[threadRow * BLOCKSIZE + threadCol] = A[threadRow * K + threadCol];
        // Bs[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];
        // __syncthreads();

        // cg::memcpy_async(
        // tile,
        // As[tile.meta_group_rank()],
        // A + (tile.meta_group_rank() * K),
        // cuda::aligned_size_t<128>(sizeof(float) * tile.size())
        // );

        // cg::memcpy_async(
        // tile,
        // Bs[tile.meta_group_rank()],
        // B + (tile.meta_group_rank() * K),
        // cuda::aligned_size_t<128>(sizeof(float) * tile.size())
        // );
        asm volatile(
          "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
          :
          : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&As[threadRow * BLOCKSIZE + threadCol]))),
            "l"(&A[(cRow * BLOCKSIZE * K) + (BLOCKSIZE * bkIdx) + threadRow * K + threadCol])
          : "memory"
        );

        asm volatile(
          "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
          :
          : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&Bs[threadRow * BLOCKSIZE + threadCol]))),
            "l"(&B[(cCol * BLOCKSIZE) + (BLOCKSIZE * N * bkIdx) + threadRow * N + threadCol])
          : "memory"
        );

        asm volatile(
          "cp.async.commit_group;\n"
        );

        asm volatile(
          "cp.async.wait_group 0;\n"
        );
        __syncthreads();
        // if(cRow == 0 && cCol == 0){
        //   printf("Tx: %d Ty: %d MR: %d\n", threadIdx.x, threadIdx.y, tile.group_meta_rank());
        // }

        // cg::wait(group);

        // block threads in this block until cache is fully populated
        // A += BLOCKSIZE;
        // B += BLOCKSIZE * N;

        // execute the dotproduct on the currently cached block
        for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
            tmp += As[threadRow * BLOCKSIZE + dotIdx] *
            Bs[dotIdx * BLOCKSIZE + threadCol];
        }
        __syncthreads();
    }
    C[threadRow * N + threadCol] = tmp;
}


__device__ __forceinline__ float compute(const float As[BLOCKSIZE][BLOCKSIZE], const float Bs[BLOCKSIZE][BLOCKSIZE], const uint threadRow, const uint threadCol){
    float tmp = 0.0f;
    for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
        tmp += As[threadRow][dotIdx] *
                Bs[dotIdx][threadCol];
    }
    return tmp;
}
  
extern "C" __global__ void sgemm_shared_mem_block_async_overlap(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C) {
    // the output block that we want to compute in this threadblock
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;
  
    // auto group = cg::this_thread_block();
    // auto tile = cg::tiled_partition<BLOCKSIZE>(group);
    // auto thread = cg::this_thread();
  
    // allocate buffer for current block in fast shared mem
    // shared mem is shared between all threads in a block
    __shared__ float As[2][BLOCKSIZE][BLOCKSIZE];
    __shared__ float Bs[2][BLOCKSIZE][BLOCKSIZE];
    // __shared__ cuda::pipeline_shared_state<cuda::thread_scope_block, 2> shared_state;
    // auto pipe = cuda::make_pipeline(group, &shared_state);
    // cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
  
  
    // the inner row & col that we're accessing in this thread
    const uint threadCol = threadIdx.x;
    const uint threadRow = threadIdx.y;
  
    // advance pointers to the starting positions
    // A += cRow * BLOCKSIZE * K;                    // row=cRow, col=0
    // B += cCol * BLOCKSIZE;                        // row=0, col=cCol
    C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol
  
    float tmp = 0.0f;
    size_t batch_sz = M / BLOCKSIZE;
    size_t stage;
  
    // cuda::memcpy_async(thread, &As[stage][threadRow][threadCol], &A[threadRow * K + threadCol], sizeof(float), pipe);
    // cuda::memcpy_async(thread, &Bs[stage][threadRow][threadCol], &B[threadRow * K + threadCol], sizeof(float), pipe);
  
    asm volatile(
      "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
      :
      : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&As[0][threadRow][threadCol]))),
        "l"(&A[(cRow * BLOCKSIZE * K) + (BLOCKSIZE * 0) + threadRow * K + threadCol])
      : "memory"
    );
  
    asm volatile(
      "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
      :
      : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&Bs[0][threadRow][threadCol]))),
        "l"(&B[(cCol * BLOCKSIZE) + (BLOCKSIZE * N * 0) + threadRow * N + threadCol])
      : "memory"
    );
  
    // pipe.producer_commit();
    asm volatile(
        "cp.async.commit_group;\n"
    );
  
    asm volatile(
      "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
      :
      : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&As[1][threadRow][threadCol]))),
        "l"(&A[(cRow * BLOCKSIZE * K) + (BLOCKSIZE * 1) + threadRow * K + threadCol])
      : "memory"
    );
  
    asm volatile(
      "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
      :
      : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&Bs[1][threadRow][threadCol]))),
        "l"(&B[(cCol * BLOCKSIZE) + (BLOCKSIZE * N * 1) + threadRow * N + threadCol])
      : "memory"
    );
  
    // pipe.producer_commit();
    asm volatile(
        "cp.async.commit_group;\n"
    );
  
  
    for(size_t compute_batch = 0;compute_batch < batch_sz;compute_batch++){
      stage = compute_batch % 2;
      asm volatile(
        "cp.async.wait_group 1;\n"
      );
      // pipe.consumer_wait();
      // printf("A[%d, %d] => %f\n", threadRow, threadCol, As[stage][threadRow][threadCol]);
      // printf("B[%d, %d] => %f\n", threadRow, threadCol, Bs[stage][threadRow][threadCol]);
  
      __syncthreads();
      tmp += compute(As[stage], Bs[stage], threadRow, threadCol);
      __syncthreads();
      // pipe.consumer_release();
  
      if(2 + compute_batch < batch_sz){
        // cuda::memcpy_async(thread, &As[stage][threadRow][threadCol], &A[threadRow * K + threadCol], sizeof(float), pipe);
        // cuda::memcpy_async(thread, &Bs[stage][threadRow][threadCol], &B[threadRow * K + threadCol], sizeof(float), pipe);
  
        asm volatile(
          "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
          :
          : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&As[stage][threadRow][threadCol]))),
            "l"(&A[(cRow * BLOCKSIZE * K) + (BLOCKSIZE * (2 + compute_batch)) + threadRow * K + threadCol])
          : "memory"
        );
  
        asm volatile(
          "cp.async.ca.shared.global [%0], [%1], 4, 4;\n"
          :
          : "r"(static_cast<std::uint32_t>(__cvta_generic_to_shared(&Bs[stage][threadRow][threadCol]))),
            "l"(&B[(cCol * BLOCKSIZE) + (BLOCKSIZE * N * (2 + compute_batch)) + threadRow * N + threadCol])
          : "memory"
        );
  
        // pipe.producer_commit();
        asm volatile(
          "cp.async.commit_group;\n"
        );
      }
    }
    C[threadRow * N + threadCol] = tmp;
}  

