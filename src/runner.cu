#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "runner.cuh"
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iomanip>

float get_sec() {
  struct timeval time;
  gettimeofday(&time, NULL);
  return (1e6 * time.tv_sec + time.tv_usec);
}

float cpu_elapsed_time(float &beg, float &end) { return 1.0e-6 * (end - beg); }

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

void CudaDeviceInfo() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
};

void randomize_matrix(float *mat, int N) {
  // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
  // precision is too low and the same random number is generated.
  struct timeval time {};
  gettimeofday(&time, nullptr);
  for (int i = 0; i < N; i++) {
    float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
    tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
    mat[i] = tmp;
  }
}

void range_init_matrix(float *mat, int N) {
  for (int i = 0; i < N; i++) {
    mat[i] = i;
  }
}

void zero_init_matrix(float *mat, int N) {
  for (int i = 0; i < N; i++) {
    mat[i] = 0.0;
  }
}

void copy_matrix(const float *src, float *dest, int N) {
  int i;
  for (i = 0; src + i && dest + i && i < N; i++)
    *(dest + i) = *(src + i);
  if (i != N)
    printf("copy failed at %d while there are %d elements in total.\n", i, N);
}

void print_matrix(const float *A, int M, int N, std::ofstream &fs) {
  int i;
  fs << std::setprecision(2)
     << std::fixed; // Set floating-point precision and fixed notation
  fs << "[";
  for (i = 0; i < M * N; i++) {
    if ((i + 1) % N == 0)
      fs << std::setw(5) << A[i]; // Set field width and write the value
    else
      fs << std::setw(5) << A[i] << ", ";
    if ((i + 1) % N == 0) {
      if (i + 1 < M * N)
        fs << ";\n";
    }
  }
  fs << "]\n";
}

bool verify_matrix(float *matRef, float *matOut, int N) {
  double diff = 0.0;
  int i;
  for (i = 0; i < N; i++) {
    diff = std::fabs(matRef[i] - matOut[i]);
    if (diff > 0.015) {
      printf("Divergence! Should %lf, Is %lff (Diff %lff) at %d\n",
             matRef[i], matOut[i], diff, i);
      return false;
    }
  }
  return true;
}

int div_ceil(int numerator, int denominator) {
  std::div_t res = std::div(numerator, denominator);
  return res.rem ? (res.quot + 1) : res.quot;
}


void run_sgemm_naive(int M, int N, int K, float alpha, float *A, float *B,
                     float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_coalesce(int M, int N, int K, float alpha, float *A, float *B,
                        float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  sgemm_global_mem_coalesce<32>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_shared_mem_block(int M, int N, int K, float alpha, float *A,
                                float *B, float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32 * 32);
  // L1 cache becomes useless, since we access GMEM only via SMEM, so we carve
  // out all of L1 to SMEM. This doesn't currently make a difference, since
  // occupancy is limited by reg and thread count, but it's good to do anyway.
  hipFuncSetAttribute(reinterpret_cast<const void*>(sgemm_shared_mem_block<32>),
                       hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);
  sgemm_shared_mem_block<32>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_shared_mem_block_async(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C){
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);

  sgemm_shared_mem_block_async<32>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_shared_mem_block_async_overlap(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C){
  constexpr const int BLOCKSIZE = 32;
  dim3 gridDim(CEIL_DIV(M, BLOCKSIZE), CEIL_DIV(N, BLOCKSIZE));
  dim3 blockDim(BLOCKSIZE, BLOCKSIZE);
  // L1 cache becomes useless, since we access GMEM only via SMEM, so we carve
  // out all of L1 to SMEM. This doesn't currently make a difference, since
  // occupancy is limited by reg and thread count, but it's good to do anyway.
  hipFuncSetAttribute(reinterpret_cast<const void*>(sgemm_shared_mem_block<BLOCKSIZE>),
                       hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);
  sgemm_shared_mem_block_async_overlap<BLOCKSIZE>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_kernel(int kernel_num, int M, int N, int K, float alpha, float *A,
                float *B, float beta, float *C) {
  switch (kernel_num) {
  case 0:
    break;
  case 1:
    run_sgemm_naive(M, N, K, alpha, A, B, beta, C);
    break;
  case 2:
    run_sgemm_coalesce(M, N, K, alpha, A, B, beta, C);
    break;
  case 3:
    run_sgemm_shared_mem_block(M, N, K, alpha, A, B, beta, C);
    break;
  case 4:
    run_sgemm_shared_mem_block_async(M, N, K, alpha, A, B, beta, C);
    break;
  case 5:
    run_sgemm_shared_mem_block_async_overlap(M, N, K, alpha, A, B, beta, C);
    break;
  default:
    throw std::invalid_argument("Unknown kernel number");
  }
}